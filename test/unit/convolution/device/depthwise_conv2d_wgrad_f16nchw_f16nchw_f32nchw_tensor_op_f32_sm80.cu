
#include <hip/hip_runtime.h>
/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 *modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice,
 *this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *notice, this list of conditions and the following disclaimer in the
 *documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its
 *contributors may be used to endorse or promote products derived from this
 *software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 *ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 *DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 *ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TOR
 *(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/**
 * \file
 * test/unit/convolution/device/depthwise_conv2d_dgrad_f32nchw_f32nchw_f32nchw_simt_f32_sm50.cu
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */
/*! \file
    \brief Tests for device-wide GEMM interface
*/
#if defined(__CUDACC__) && (__CUDACC_VER_MAJOR__ >= 11)
#include "cutlass/convolution/device/convolution.h"

#include "../../common/cutlass_unit_test.h"

#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/tensor_view_io.h"

#include "conv2d_wgrad_testbed.h"

#define RUN_DEPTHWISE_CONVOLUTION(stage)                                       \
    do {                                                                       \
        using ElementOutput = float;                                           \
        using ElementAccumulator = float;                                      \
        using ElementCompute = float;                                          \
        using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;          \
        using Convolution = cutlass::conv::device::ConvolutionBackwardFilter<  \
                cutlass::half_t, cutlass::layout::TensorNCHW, cutlass::half_t, \
                cutlass::layout::TensorNCHW, ElementOutput,                    \
                cutlass::layout::TensorNCHW, ElementAccumulator,               \
                cutlass::conv::ConvType::kDepthwiseConvolution,                \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,           \
                ThreadBlockShape, WarpShape, InstructionShape,                 \
                cutlass::epilogue::thread::LinearCombination<                  \
                        ElementOutput, 1, ElementAccumulator, ElementCompute>, \
                cutlass::conv::threadblock::                                   \
                        DepthwiseConvolutionWgradThreadblockSwizzle,           \
                stage, 1, 1, cutlass::conv::SpecialOptimizeDesc::NONE,         \
                cutlass::arch::OpMultiplyAdd,                                  \
                cutlass::conv::ImplicitGemmMode::GEMM_NT>;                     \
        EXPECT_TRUE(test::convolution::device::TestDepthwiseConv2dWgrad<       \
                    Convolution>());                                           \
    } while (0)

////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     128x256x64_64x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 256, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     256x128x64_64x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<256, 128, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     128x128x64_64x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 128, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     64x128x64_32x64x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 128, 64>;
    using WarpShape = cutlass::gemm::GemmShape<32, 64, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     128x64x64_64x32x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 64, 64>;
    using WarpShape = cutlass::gemm::GemmShape<64, 32, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     64x64x64_32x32x64) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 64, 64>;
    using WarpShape = cutlass::gemm::GemmShape<32, 32, 64>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     128x256x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 256, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     256x128x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<256, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     128x128x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     256x64x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<256, 64, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     64x256x32_64x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 256, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     64x128x32_32x64x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<32, 64, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     128x64x32_64x32x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<128, 64, 32>;
    using WarpShape = cutlass::gemm::GemmShape<64, 32, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

TEST(SM80_Device_Depthwise_Conv2dWgrad_f16_f16_NCHW_tensor_op_f32,
     64x64x32_32x32x32) {
    using ThreadBlockShape = cutlass::gemm::GemmShape<64, 64, 32>;
    using WarpShape = cutlass::gemm::GemmShape<32, 32, 32>;
    RUN_DEPTHWISE_CONVOLUTION(1);
    RUN_DEPTHWISE_CONVOLUTION(2);
}

//////////////////////////////////////////////////////////////////////////////////
#endif
